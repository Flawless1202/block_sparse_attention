#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <exception>

template <typename scalar_t>
__global__ void matmul_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> a,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> b,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> out,
    const int num_heads,
    const int m,
    const int n,
    const int k) {
    const int bidx = blockIdx.z;
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < n) {
        for (int hidx = 0; hidx < num_heads; hidx++) {
            scalar_t val = 0.0;
            scalar_t y = 0.0;
            for (int i = 0; i < k; i++) {
                val += a[bidx][hidx][row][i] * b[bidx][hidx][i][col];
                // y -= a[bidx][hidx][row][i] * b[bidx][hidx][i][col];
                // scalar_t r = val - y;
                // y = (r - val) + y;
                // val = r;
            }
            out[bidx][hidx][row][col] = val;
        }
    }
}

template <typename scalar_t>
__global__ void matmul_shared_memory_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> a,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> b,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> out,
    const int num_heads,
    const int m,
    const int n,
    const int k,
    const int block_size) {
    const int bidx = blockIdx.z;
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    // __shared__ scalar_t _sh_tile_a[block_size][block_size];
    // __shared__ scalar_t _sh_tile_b[block_size][block_size];

    extern __shared__ char data[];
    scalar_t* _sh_tile_a = (scalar_t*)data;
    scalar_t* _sh_tile_b = (scalar_t*)(block_size * block_size * sizeof(scalar_t) + data);

    const int num_blocks = (k + block_size - 1) / block_size;

    if (row < m && col < n) {
        for (int hidx = 0; hidx < num_heads; hidx++) {
            scalar_t val = 0.0;
            for (int i = 0; i < num_blocks; i++) {
                _sh_tile_a[threadIdx.x * block_size + threadIdx.y] = a[bidx][hidx][row][i * block_size + threadIdx.y];
                _sh_tile_b[threadIdx.x * block_size + threadIdx.y] = b[bidx][hidx][i * block_size + threadIdx.x][col];
                __syncthreads();

                for (int j = 0; j < block_size; j++) {
                    val += _sh_tile_a[threadIdx.x * block_size + j] * _sh_tile_b[j * block_size + threadIdx.y];
                }
                __syncthreads();
            }
            out[bidx][hidx][row][col] = val;
        }
    }
}

torch::Tensor matmul_cuda(torch::Tensor a, torch::Tensor b) {
    const int bs = a.size(0);
    const int h = a.size(1);

    // the tensor a is of size `(bs, h, ma, ka)`
    const int ma = a.size(-2);
    const int ka = a.size(-1);

    // the tensor b is of size `(bs, h, kb, nb)`
    const int kb = b.size(-2);
    const int nb = b.size(-1);

    if (ka != kb) {
        throw std::invalid_argument("Size of tensor A must match size of tensor B.");
    }

    // configure cuda
    const int threads = 32;
    const dim3 threads_per_block(threads, threads, 1);
    const dim3 blocks_per_grid(ma / threads + 1, nb / threads + 1, bs);

    auto tensor_options = torch::TensorOptions().dtype(a.dtype()).device(torch::kCUDA, a.device().index());
    auto out = torch::zeros({bs, h, ma, nb}, tensor_options);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        a.type(), "matmul_cuda", ([&] {
            matmul_cuda_kernel<scalar_t><<<blocks_per_grid, threads_per_block>>>(
                a.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                b.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                out.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(), h, ma, nb, ka);
        }));
    return out;
}

torch::Tensor matmul_shared_memory_cuda(torch::Tensor a, torch::Tensor b, const int block_size = 16) {
    const int bs = a.size(0);
    const int h = a.size(1);

    // the tensor a is of size `(bs, h, ma, ka)`
    const int ma = a.size(-2);
    const int ka = a.size(-1);

    // the tensor b is of size `(bs, h, kb, nb)`
    const int kb = b.size(-2);
    const int nb = b.size(-1);

    if (ka != kb) {
        throw std::invalid_argument("Size of tensor A must match size of tensor B.");
    }

    // configure cuda
    const dim3 threads_per_block(block_size, block_size, 1);
    const dim3 blocks_per_grid(ma / block_size + 1, nb / block_size + 1, bs);

    auto tensor_options = torch::TensorOptions().dtype(a.dtype()).device(torch::kCUDA, a.device().index());
    auto out = torch::zeros({bs, h, ma, nb}, tensor_options);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        a.type(), "matmul_shared_memory_cuda", ([&] {
            matmul_shared_memory_cuda_kernel<scalar_t>
                <<<blocks_per_grid, threads_per_block, 2 * block_size * block_size * sizeof(scalar_t)>>>(
                    a.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                    b.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                    out.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(), h, ma, nb, ka, block_size);
        }));
    return out;
}
